#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>

// Benchmark parameters
#define MB_TO_TEST 1024  // Test 1GB of data
#define BLOCK_SIZE 16    // AES block size in bytes
// #define NUM_BLOCKS ((MB_TO_TEST * 1024 * 1024) / BLOCK_SIZE)
#define THREADS_PER_BLOCK 256
uint64_t NUM_BLOCKS = 128 * 1024 * 1024;

// AES parameters
#define Nb 4
#define Nk 4
#define Nr 10

// Error checking macro
#define CHECK_CUDA_ERROR(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", \
                __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

// Device constant memory for tables
__constant__ uint8_t d_sbox[256];
__constant__ uint8_t d_mul2[256];
__constant__ uint8_t d_mul3[256];
__constant__ uint8_t d_Rcon[11];

typedef struct {
    uint8_t nonce[8];    // 64-bit nonce
    uint8_t counter[8];  // 64-bit counter
} ctr_block_t;

// Host-side tables
static const uint8_t h_sbox[256] = {
    0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
    0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
    0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
    0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
    0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
    0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
    0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
    0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
    0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
    0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
    0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
    0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
    0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
    0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
    0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
    0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
};

// Host-side multiplication tables
static const uint8_t h_mul2[256] = {
    0x00, 0x02, 0x04, 0x06, 0x08, 0x0a, 0x0c, 0x0e, 0x10, 0x12, 0x14, 0x16, 0x18, 0x1a, 0x1c, 0x1e,
    0x20, 0x22, 0x24, 0x26, 0x28, 0x2a, 0x2c, 0x2e, 0x30, 0x32, 0x34, 0x36, 0x38, 0x3a, 0x3c, 0x3e,
    0x40, 0x42, 0x44, 0x46, 0x48, 0x4a, 0x4c, 0x4e, 0x50, 0x52, 0x54, 0x56, 0x58, 0x5a, 0x5c, 0x5e,
    0x60, 0x62, 0x64, 0x66, 0x68, 0x6a, 0x6c, 0x6e, 0x70, 0x72, 0x74, 0x76, 0x78, 0x7a, 0x7c, 0x7e,
    0x80, 0x82, 0x84, 0x86, 0x88, 0x8a, 0x8c, 0x8e, 0x90, 0x92, 0x94, 0x96, 0x98, 0x9a, 0x9c, 0x9e,
    0xa0, 0xa2, 0xa4, 0xa6, 0xa8, 0xaa, 0xac, 0xae, 0xb0, 0xb2, 0xb4, 0xb6, 0xb8, 0xba, 0xbc, 0xbe,
    0xc0, 0xc2, 0xc4, 0xc6, 0xc8, 0xca, 0xcc, 0xce, 0xd0, 0xd2, 0xd4, 0xd6, 0xd8, 0xda, 0xdc, 0xde,
    0xe0, 0xe2, 0xe4, 0xe6, 0xe8, 0xea, 0xec, 0xee, 0xf0, 0xf2, 0xf4, 0xf6, 0xf8, 0xfa, 0xfc, 0xfe,
    0x1b, 0x19, 0x1f, 0x1d, 0x13, 0x11, 0x17, 0x15, 0x0b, 0x09, 0x0f, 0x0d, 0x03, 0x01, 0x07, 0x05,
    0x3b, 0x39, 0x3f, 0x3d, 0x33, 0x31, 0x37, 0x35, 0x2b, 0x29, 0x2f, 0x2d, 0x23, 0x21, 0x27, 0x25,
    0x5b, 0x59, 0x5f, 0x5d, 0x53, 0x51, 0x57, 0x55, 0x4b, 0x49, 0x4f, 0x4d, 0x43, 0x41, 0x47, 0x45,
    0x7b, 0x79, 0x7f, 0x7d, 0x73, 0x71, 0x77, 0x75, 0x6b, 0x69, 0x6f, 0x6d, 0x63, 0x61, 0x67, 0x65,
    0x9b, 0x99, 0x9f, 0x9d, 0x93, 0x91, 0x97, 0x95, 0x8b, 0x89, 0x8f, 0x8d, 0x83, 0x81, 0x87, 0x85,
    0xbb, 0xb9, 0xbf, 0xbd, 0xb3, 0xb1, 0xb7, 0xb5, 0xab, 0xa9, 0xaf, 0xad, 0xa3, 0xa1, 0xa7, 0xa5,
    0xdb, 0xd9, 0xdf, 0xdd, 0xd3, 0xd1, 0xd7, 0xd5, 0xcb, 0xc9, 0xcf, 0xcd, 0xc3, 0xc1, 0xc7, 0xc5,
    0xfb, 0xf9, 0xff, 0xfd, 0xf3, 0xf1, 0xf7, 0xf5, 0xeb, 0xe9, 0xef, 0xed, 0xe3, 0xe1, 0xe7, 0xe5
};
static const uint8_t h_mul3[256] = {
    0x00, 0x03, 0x06, 0x05, 0x0c, 0x0f, 0x0a, 0x09, 0x18, 0x1b, 0x1e, 0x1d, 0x14, 0x17, 0x12, 0x11,
    0x30, 0x33, 0x36, 0x35, 0x3c, 0x3f, 0x3a, 0x39, 0x28, 0x2b, 0x2e, 0x2d, 0x24, 0x27, 0x22, 0x21,
    0x60, 0x63, 0x66, 0x65, 0x6c, 0x6f, 0x6a, 0x69, 0x78, 0x7b, 0x7e, 0x7d, 0x74, 0x77, 0x72, 0x71,
    0x50, 0x53, 0x56, 0x55, 0x5c, 0x5f, 0x5a, 0x59, 0x48, 0x4b, 0x4e, 0x4d, 0x44, 0x47, 0x42, 0x41,
    0xc0, 0xc3, 0xc6, 0xc5, 0xcc, 0xcf, 0xca, 0xc9, 0xd8, 0xdb, 0xde, 0xdd, 0xd4, 0xd7, 0xd2, 0xd1,
    0xf0, 0xf3, 0xf6, 0xf5, 0xfc, 0xff, 0xfa, 0xf9, 0xe8, 0xeb, 0xee, 0xed, 0xe4, 0xe7, 0xe2, 0xe1,
    0xa0, 0xa3, 0xa6, 0xa5, 0xac, 0xaf, 0xaa, 0xa9, 0xb8, 0xbb, 0xbe, 0xbd, 0xb4, 0xb7, 0xb2, 0xb1,
    0x90, 0x93, 0x96, 0x95, 0x9c, 0x9f, 0x9a, 0x99, 0x88, 0x8b, 0x8e, 0x8d, 0x84, 0x87, 0x82, 0x81,
    0x9b, 0x98, 0x9d, 0x9e, 0x97, 0x94, 0x91, 0x92, 0x83, 0x80, 0x85, 0x86, 0x8f, 0x8c, 0x89, 0x8a,
    0xab, 0xa8, 0xad, 0xae, 0xa7, 0xa4, 0xa1, 0xa2, 0xb3, 0xb0, 0xb5, 0xb6, 0xbf, 0xbc, 0xb9, 0xba,
    0xfb, 0xf8, 0xfd, 0xfe, 0xf7, 0xf4, 0xf1, 0xf2, 0xe3, 0xe0, 0xe5, 0xe6, 0xef, 0xec, 0xe9, 0xea,
    0xcb, 0xc8, 0xcd, 0xce, 0xc7, 0xc4, 0xc1, 0xc2, 0xd3, 0xd0, 0xd5, 0xd6, 0xdf, 0xdc, 0xd9, 0xda,
    0x5b, 0x58, 0x5d, 0x5e, 0x57, 0x54, 0x51, 0x52, 0x43, 0x40, 0x45, 0x46, 0x4f, 0x4c, 0x49, 0x4a,
    0x6b, 0x68, 0x6d, 0x6e, 0x67, 0x64, 0x61, 0x62, 0x73, 0x70, 0x75, 0x76, 0x7f, 0x7c, 0x79, 0x7a,
    0x3b, 0x38, 0x3d, 0x3e, 0x37, 0x34, 0x31, 0x32, 0x23, 0x20, 0x25, 0x26, 0x2f, 0x2c, 0x29, 0x2a,
    0x0b, 0x08, 0x0d, 0x0e, 0x07, 0x04, 0x01, 0x02, 0x13, 0x10, 0x15, 0x16, 0x1f, 0x1c, 0x19, 0x1a
};
static const uint8_t h_Rcon[11] = {
    0x00, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1B, 0x36
};

__host__ void increment_counter(uint8_t* counter) {
    for (int i = 7; i >= 0; i--) {
        if (++counter[i] != 0) break;
    }
}

__device__ void increment_counter_device(uint8_t* counter) {
    for (int i = 7; i >= 0; i--) {
        if (++counter[i] != 0) break;
    }
}

__device__ void set_counter_value(uint8_t* counter, uint64_t value) {
    counter[7] = value & 0xFF;
    counter[6] = (value >> 8) & 0xFF;
    counter[5] = (value >> 16) & 0xFF;
    counter[4] = (value >> 24) & 0xFF;
    counter[3] = (value >> 32) & 0xFF;
    counter[2] = (value >> 40) & 0xFF;
    counter[1] = (value >> 48) & 0xFF;
    counter[0] = (value >> 56) & 0xFF;
}

// Modified CUDA kernel for CTR mode
__global__ void AES_Encrypt_CTR_Kernel(uint8_t* input, const uint8_t* roundKey, 
                                     uint8_t* output, int num_blocks, 
                                     const uint8_t* base_nonce, const uint8_t* base_counter) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= num_blocks) return;

    // Create local counter block
    uint8_t counter_block[16];
    
    // Copy nonce (first 8 bytes)
    for (int i = 0; i < 8; i++) {
        counter_block[i] = base_nonce[i];
    }
    
    // Calculate counter value directly
    uint64_t base_counter_value = 0;
    for (int i = 0; i < 8; i++) {
        base_counter_value = (base_counter_value << 8) | base_counter[i];
    }
    uint64_t new_counter_value = base_counter_value + idx;
    set_counter_value(counter_block + 8, new_counter_value);

    // Create state array for AES operation
    uint8_t state[4][4];
    uint8_t keystream[16];

    // Copy counter block to state array
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            state[j][i] = counter_block[i * 4 + j];
        }
    }

    // Initial round
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            state[j][i] ^= roundKey[i * 4 + j];
        }
    }

    // Main rounds
    for (int round = 1; round < Nr; round++) {
        // SubBytes
        for (int i = 0; i < 4; i++) {
            for (int j = 0; j < 4; j++) {
                state[i][j] = d_sbox[state[i][j]];
            }
        }

        // ShiftRows
        uint8_t temp;
        temp = state[1][0];
        state[1][0] = state[1][1];
        state[1][1] = state[1][2];
        state[1][2] = state[1][3];
        state[1][3] = temp;

        temp = state[2][0];
        state[2][0] = state[2][2];
        state[2][2] = temp;
        temp = state[2][1];
        state[2][1] = state[2][3];
        state[2][3] = temp;

        temp = state[3][3];
        state[3][3] = state[3][2];
        state[3][2] = state[3][1];
        state[3][1] = state[3][0];
        state[3][0] = temp;

        // MixColumns
        uint8_t temp_state[4][4];
        for (int i = 0; i < 4; i++) {
            for (int j = 0; j < 4; j++) {
                temp_state[j][i] = state[j][i];
            }
        }

        for (int i = 0; i < 4; i++) {
            state[0][i] = d_mul2[temp_state[0][i]] ^ d_mul3[temp_state[1][i]] ^ 
                         temp_state[2][i] ^ temp_state[3][i];
            state[1][i] = temp_state[0][i] ^ d_mul2[temp_state[1][i]] ^ 
                         d_mul3[temp_state[2][i]] ^ temp_state[3][i];
            state[2][i] = temp_state[0][i] ^ temp_state[1][i] ^ 
                         d_mul2[temp_state[2][i]] ^ d_mul3[temp_state[3][i]];
            state[3][i] = d_mul3[temp_state[0][i]] ^ temp_state[1][i] ^ 
                         temp_state[2][i] ^ d_mul2[temp_state[3][i]];
        }

        // AddRoundKey
        for (int i = 0; i < 4; i++) {
            for (int j = 0; j < 4; j++) {
                state[j][i] ^= roundKey[round * 16 + i * 4 + j];
            }
        }
    }

    // Final round
    // SubBytes
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            state[i][j] = d_sbox[state[i][j]];
        }
    }

    // ShiftRows
    uint8_t temp;
    temp = state[1][0];
    state[1][0] = state[1][1];
    state[1][1] = state[1][2];
    state[1][2] = state[1][3];
    state[1][3] = temp;

    temp = state[2][0];
    state[2][0] = state[2][2];
    state[2][2] = temp;
    temp = state[2][1];
    state[2][1] = state[2][3];
    state[2][3] = temp;

    temp = state[3][3];
    state[3][3] = state[3][2];
    state[3][2] = state[3][1];
    state[3][1] = state[3][0];
    state[3][0] = temp;

    // AddRoundKey
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            state[j][i] ^= roundKey[Nr * 16 + i * 4 + j];
        }
    }

    // Copy state to keystream
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            keystream[i * 4 + j] = state[j][i];
        }
    }

    // XOR keystream with input to create output
    uint8_t* block_input = input + (idx * 16);
    uint8_t* block_output = output + (idx * 16);
    for (int i = 0; i < 16; i++) {
        block_output[i] = block_input[i] ^ keystream[i];
    }
}

void KeyExpansion(uint8_t* key, uint8_t* roundKey) {
    uint8_t temp[4];
    
    // First round key is the original key
    for(int i = 0; i < Nk * 4; i++) {
        roundKey[i] = key[i];
    }
    
    // Generate round keys
    int i = Nk;
    while(i < Nb * (Nr + 1)) {
        // Copy last 4 bytes to temp
        for(int j = 0; j < 4; j++) {
            temp[j] = roundKey[(i - 1) * 4 + j];
        }
        
        if(i % Nk == 0) {
            // Rotate word
            uint8_t k = temp[0];
            temp[0] = temp[1];
            temp[1] = temp[2];
            temp[2] = temp[3];
            temp[3] = k;
            
            // Apply S-box
            for(int j = 0; j < 4; j++) {
                temp[j] = h_sbox[temp[j]];
            }
            
            // XOR with round constant
            temp[0] = temp[0] ^ h_Rcon[i/Nk];
        }
        
        // XOR with previous round key
        for(int j = 0; j < 4; j++) {
            roundKey[i * 4 + j] = roundKey[(i - Nk) * 4 + j] ^ temp[j];
        }
        i++;
    }
}

// Modified host function to run AES-CTR encryption on GPU
void AES_Encrypt_CTR_CUDA(uint8_t* input, uint8_t* key, uint8_t* output, 
                         int num_blocks, ctr_block_t* initial_ctr) {
    uint8_t roundKey[176];  // 11 round keys (176 bytes)
    KeyExpansion(key, roundKey);

    // Allocate device memory
    uint8_t *d_input, *d_output, *d_roundKey, *d_nonce, *d_counter;
    
    CHECK_CUDA_ERROR(hipMalloc(&d_input, num_blocks * 16));
    CHECK_CUDA_ERROR(hipMalloc(&d_output, num_blocks * 16));
    CHECK_CUDA_ERROR(hipMalloc(&d_roundKey, 176));
    CHECK_CUDA_ERROR(hipMalloc(&d_nonce, 8));
    CHECK_CUDA_ERROR(hipMalloc(&d_counter, 8));

    // Copy data to device
    CHECK_CUDA_ERROR(hipMemcpy(d_input, input, num_blocks * 16, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_roundKey, roundKey, 176, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_nonce, initial_ctr->nonce, 8, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_counter, initial_ctr->counter, 8, hipMemcpyHostToDevice));

    // Copy lookup tables to constant memory
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_sbox), h_sbox, sizeof(h_sbox)));
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_mul2), h_mul2, sizeof(h_mul2)));
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_mul3), h_mul3, sizeof(h_mul3)));

    // Calculate grid dimensions
    int threadsPerBlock = THREADS_PER_BLOCK;
    int blocksPerGrid = (num_blocks + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel
    AES_Encrypt_CTR_Kernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_input, d_roundKey, d_output, num_blocks, d_nonce, d_counter);
    CHECK_CUDA_ERROR(hipGetLastError());

    // Copy result back to host
    CHECK_CUDA_ERROR(hipMemcpy(output, d_output, num_blocks * 16, hipMemcpyDeviceToHost));

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_roundKey);
    hipFree(d_nonce);
    hipFree(d_counter);
}


void AES_Encrypt_Serial_CTR(uint8_t* input, uint8_t* key, uint8_t* output, 
                           int num_blocks, ctr_block_t* initial_ctr) {
    uint8_t roundKey[176];
    KeyExpansion(key, roundKey);
    
    uint8_t counter_block[16];
    uint8_t keystream[16];
    ctr_block_t current_ctr = *initial_ctr;
    
    for(int block = 0; block < num_blocks; block++) {
        // Set up counter block
        memcpy(counter_block, current_ctr.nonce, 8);
        memcpy(counter_block + 8, current_ctr.counter, 8);
        
        // Encrypt counter block to generate keystream
        uint8_t state[4][4];
        for(int i = 0; i < 4; i++) {
            for(int j = 0; j < 4; j++) {
                state[j][i] = counter_block[i * 4 + j];
            }
        }
        
        // Initial round
        for(int i = 0; i < 4; i++) {
            for(int j = 0; j < 4; j++) {
                state[j][i] ^= roundKey[i * 4 + j];
            }
        }
        
        // Main rounds
        for(int round = 1; round < Nr; round++) {
            // SubBytes
            for(int i = 0; i < 4; i++) {
                for(int j = 0; j < 4; j++) {
                    state[i][j] = h_sbox[state[i][j]];
                }
            }
            
            // ShiftRows
            uint8_t temp;
            temp = state[1][0];
            state[1][0] = state[1][1];
            state[1][1] = state[1][2];
            state[1][2] = state[1][3];
            state[1][3] = temp;
            
            temp = state[2][0];
            state[2][0] = state[2][2];
            state[2][2] = temp;
            temp = state[2][1];
            state[2][1] = state[2][3];
            state[2][3] = temp;
            
            temp = state[3][3];
            state[3][3] = state[3][2];
            state[3][2] = state[3][1];
            state[3][1] = state[3][0];
            state[3][0] = temp;
            
            // MixColumns
            uint8_t temp_state[4][4];
            for(int i = 0; i < 4; i++) {
                for(int j = 0; j < 4; j++) {
                    temp_state[j][i] = state[j][i];
                }
            }
            
            for(int i = 0; i < 4; i++) {
                state[0][i] = h_mul2[temp_state[0][i]] ^ h_mul3[temp_state[1][i]] ^ 
                             temp_state[2][i] ^ temp_state[3][i];
                state[1][i] = temp_state[0][i] ^ h_mul2[temp_state[1][i]] ^ 
                             h_mul3[temp_state[2][i]] ^ temp_state[3][i];
                state[2][i] = temp_state[0][i] ^ temp_state[1][i] ^ 
                             h_mul2[temp_state[2][i]] ^ h_mul3[temp_state[3][i]];
                state[3][i] = h_mul3[temp_state[0][i]] ^ temp_state[1][i] ^ 
                             temp_state[2][i] ^ h_mul2[temp_state[3][i]];
            }
            
            // AddRoundKey
            for(int i = 0; i < 4; i++) {
                for(int j = 0; j < 4; j++) {
                    state[j][i] ^= roundKey[round * 16 + i * 4 + j];
                }
            }
        }
        
        // Final round
        // SubBytes
        for(int i = 0; i < 4; i++) {
            for(int j = 0; j < 4; j++) {
                state[i][j] = h_sbox[state[i][j]];
            }
        }
        
        // ShiftRows
        uint8_t temp;
        temp = state[1][0];
        state[1][0] = state[1][1];
        state[1][1] = state[1][2];
        state[1][2] = state[1][3];
        state[1][3] = temp;
        
        temp = state[2][0];
        state[2][0] = state[2][2];
        state[2][2] = temp;
        temp = state[2][1];
        state[2][1] = state[2][3];
        state[2][3] = temp;
        
        temp = state[3][3];
        state[3][3] = state[3][2];
        state[3][2] = state[3][1];
        state[3][1] = state[3][0];
        state[3][0] = temp;
        
        // AddRoundKey
        for(int i = 0; i < 4; i++) {
            for(int j = 0; j < 4; j++) {
                state[j][i] ^= roundKey[Nr * 16 + i * 4 + j];
            }
        }

        // Copy state to keystream
        for(int i = 0; i < 4; i++) {
            for(int j = 0; j < 4; j++) {
                keystream[i * 4 + j] = state[j][i];
            }
        }

        // XOR keystream with input to get output
        uint8_t* block_input = input + (block * 16);
        uint8_t* block_output = output + (block * 16);
        for(int i = 0; i < 16; i++) {
            block_output[i] = block_input[i] ^ keystream[i];
        }
        
        // Increment counter
        increment_counter(current_ctr.counter);
    }
}

// Function to compare two buffers
int compare_buffers(uint8_t* buf1, uint8_t* buf2, size_t size) {
    for(size_t i = 0; i < size; i++) {
        if(buf1[i] != buf2[i]) {
            printf("Mismatch at position %zu: %02x != %02x\n", i, buf1[i], buf2[i]);
            return 0;
        }
    }
    return 1;
}

int main() {
    uint8_t key[16] = {
        0x2b, 0x7e, 0x15, 0x16,
        0x28, 0xae, 0xd2, 0xa6,
        0xab, 0xf7, 0x15, 0x88,
        0x09, 0xcf, 0x4f, 0x3c
    };
    ctr_block_t initial_ctr = {
        .nonce = {0x01, 0x23, 0x45, 0x67, 0x89, 0xAB, 0xCD, 0xEF},
        .counter = {0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00}
    };
    
    // Allocate host memory with page-locked memory for better transfer performance
    uint8_t *input, *output_gpu, *output_cpu;
    CHECK_CUDA_ERROR(hipHostMalloc(&input, NUM_BLOCKS * 16, hipHostMallocDefault));
    CHECK_CUDA_ERROR(hipHostMalloc(&output_gpu, NUM_BLOCKS * 16, hipHostMallocDefault));
    CHECK_CUDA_ERROR(hipHostMalloc(&output_cpu, NUM_BLOCKS * 16, hipHostMallocDefault));
    
    // Initialize input data
    for(int i = 0; i < NUM_BLOCKS * 16; i++) {
        input[i] = i & 0xFF;
    }
    
    // Get device properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Using GPU: %s\n", prop.name);
    
    AES_Encrypt_CTR_CUDA(input, key, output_gpu, NUM_BLOCKS, &initial_ctr);
    
    // Benchmark GPU version
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    AES_Encrypt_CTR_CUDA(input, key, output_gpu, NUM_BLOCKS, &initial_ctr);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float gpu_milliseconds = 0;
    hipEventElapsedTime(&gpu_milliseconds, start, stop);
    double gpu_seconds = gpu_milliseconds / 1000.0;
    
    // Benchmark CPU version
    clock_t cpu_start = clock();
    AES_Encrypt_Serial_CTR(input, key, output_cpu, NUM_BLOCKS, &initial_ctr);
    clock_t cpu_end = clock();
    double cpu_seconds = (double)(cpu_end - cpu_start) / CLOCKS_PER_SEC;
    
    // Compare results
    printf("Results match: %s\n", 
           compare_buffers(output_cpu, output_gpu, NUM_BLOCKS * 16) ? "Yes" : "No");
    
    // Print first block comparison if there's a mismatch
    if (!compare_buffers(output_cpu, output_gpu, NUM_BLOCKS * 16)) {
        printf("\nFirst block comparison:\nCPU: ");
        for(int i = 0; i < 16; i++) {
            printf("%02x ", output_cpu[i]);
        }
        printf("\nGPU: ");
        for(int i = 0; i < 16; i++) {
            printf("%02x ", output_gpu[i]);
        }
        printf("\n");
    }
    
    // Calculate and print performance metrics
    double data_size_gb = (double)(NUM_BLOCKS * BLOCK_SIZE) / (1024 * 1024 * 1024);
    printf("\nPerformance Comparison:\n");
    printf("Data size: %.2f GB\n", data_size_gb);
    printf("CPU Time: %.4f seconds (%.2f GB/s)\n", 
           cpu_seconds, data_size_gb / cpu_seconds);
    printf("GPU Time: %.4f seconds (%.2f GB/s)\n", 
           gpu_seconds, data_size_gb / gpu_seconds);
    printf("Speedup: %.2fx\n", cpu_seconds / gpu_seconds);
    
    // Clean up
    hipHostFree(input);
    hipHostFree(output_gpu);
    hipHostFree(output_cpu);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}